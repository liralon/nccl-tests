/*************************************************************************
 * Copyright (c) 2024, AMAZON CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

#define ALIGN 4

void AllGatherMeshCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  size_t base = (count/(ALIGN*nranks))*ALIGN;
  *sendcount = base;
  *recvcount = base*nranks;
  *sendInplaceOffset = base;
  *recvInplaceOffset = 0;
  *paramcount = base;
}

testResult_t AllGatherMeshInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;

  for (int i=0; i<args->nGpus; i++) {
    CUDACHECK(hipSetDevice(args->gpus[i]));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? ((char*)args->recvbuffs[i])+rank*args->sendBytes : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, 0, type, ncclSum, 33*rep + rank, 1, 0));
    for (int j=0; j<nranks; j++) {
      TESTCHECK(InitData((char*)args->expected[i] + args->sendBytes*j, sendcount, 0, type, ncclSum, 33*rep + j, 1, 0));
    }
    CUDACHECK(hipDeviceSynchronize());
  }
  return testSuccess;
}

void AllGatherMeshGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {

  double baseBw = (double)(count * typesize * nranks) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = ((double)(nranks - 1))/((double)nranks);
  *busBw = baseBw * factor;
}

testResult_t AllGatherMeshRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  char* rbuff = (char*)recvbuff;
  int nRanks;
  NCCLCHECK(ncclCommCount(comm, &nRanks));
  int rank;
  NCCLCHECK(ncclCommUserRank(comm, &rank));
  size_t rankSize = count * wordSize(type);

  NCCLCHECK(ncclGroupStart());
  for (int peerRank = 0; peerRank < nRanks; peerRank++) {
    if (peerRank == rank) {
      continue;
    }
    NCCLCHECK(ncclSend(sendbuff, count, type, peerRank, comm, stream));
    NCCLCHECK(ncclRecv(rbuff + peerRank*rankSize, count, type, peerRank, comm, stream));
  }
  NCCLCHECK(ncclGroupEnd());

  return testSuccess;
}

struct testColl allGatherMeshTest = {
  "AllGatherMesh",
  AllGatherMeshCollByteCount,
  AllGatherMeshInitData,
  AllGatherMeshGetBw,
  AllGatherMeshRunColl
};

void AllGatherMeshGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  AllGatherMeshCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t AllGatherMeshRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &allGatherMeshTest;
  ncclDataType_t *run_types;
  const char **run_typenames;
  int type_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  for (int i=0; i<type_count; i++) {
    TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "none", -1));
  }

  return testSuccess;
}

struct testEngine allGatherMeshEngine = {
  AllGatherMeshGetBuffSize,
  AllGatherMeshRunTest
};

#pragma weak ncclTestEngine=allGatherMeshEngine
